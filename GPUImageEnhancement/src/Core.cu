#include "Core.cuh"

using namespace gpu_enhance;

bool gpue_CUDA_CheckError(hipError_t status, const char* WhoCalled, const char* WhatCalled) 
{
	if (status != hipSuccess)
	{
		std::cerr << WhoCalled << ": " << WhatCalled << " failed! CudaStatus: " << hipGetErrorString(status) << std::endl;
		return true;
	}
	return false;
}

bool gpue_CUDA_AllocateOnHost(void** ptr, size_t size, unsigned int flags)
{
	hipError_t status = hipHostAlloc(ptr, size, flags);
	if (gpue_CUDA_CheckError(status, "gpue_CUDA_AllocateOnHost", "hipHostAlloc"))
	{
		hipFree(ptr);
		return false;
	}

	return true;
}

bool gpue_CUDA_AllocateOnDevice(void** ptr, size_t size, unsigned int flags)
{
	hipError_t status = hipMallocManaged(ptr, size, flags);
	if (gpue_CUDA_CheckError(status, "gpue_CUDA_AllocateOnDevice", "hipMallocManaged"))
	{
		hipFree(ptr);
		return false;
	}

	return true;
}

bool gpue_CUDA_FreeFromDevice(void* dev_ptr)
{
	hipError_t status = hipFree(dev_ptr);
	return !gpue_CUDA_CheckError(status, "gpue_CUDA_FreeFromDevice", "hipFree");
}

bool gpue_CUDA_FreeFromHost(void* host_ptr)
{
	hipError_t status = hipHostFree(host_ptr);
	return !gpue_CUDA_CheckError(status, "gpue_CUDA_FreeFromHost", "hipHostFree");;
}

bool gpue_CUDA_Memcpy(void* dst, const void* src, size_t count, hipMemcpyKind kind)
{
	hipError_t status = hipMemcpy(dst, src, count, kind);
	return !gpue_CUDA_CheckError(status, "gpue_CUDA_Memcpy", "hipMemcpy");
}